#include "hip/hip_runtime.h"
/**
 * @file tensorgpufunction.cu
 * @brief Implements GPU functions for each operation.
 * 
 * @author Zoe Lurie
 * @date November 2024
 */

#include "tensorgpufunctions.h"

#define NUMBLOCKS 256
#define NUMTHREADS 256
#define NUMBLOCKS2D 16
#define NUMTHREADS2D 16
#define NUMBLOCKS3D 8
#define NUMTHREADS3D 8

dim3 numblocks2dDIM(NUMBLOCKS2D, NUMBLOCKS2D);
dim3 numthreads2dDIM(NUMTHREADS2D, NUMTHREADS2D);
dim3 numblocks3dDIM(NUMBLOCKS3D, NUMBLOCKS3D, NUMBLOCKS3D);
dim3 numthreads3dDIM(NUMTHREADS3D, NUMTHREADS3D, NUMTHREADS3D);


__global__ void gpuNeg(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = -data1[i];
    }
}

__global__ void gpuAdd(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] + data2[i];
    }
}

__global__ void gpuAddScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] + n;
    }
}

__global__ void gpuSubtract(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] - data2[i];
    }
}

__global__ void gpuSubtractScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] - n;
    }
}

__global__ void gpuScalarSubtract(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n - data1[i];
    }
}

__global__ void gpuPow(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = pow(data1[i], n);
    }
}

__global__ void gpuZeroes(double * ret, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = 0;
    }
}

__global__ void gpuOnes(double * ret, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = 1;
    }
}

__global__ void gpuFill(double * ret, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n;
    }
}

__global__ void gpuElementwiseMult(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] * data2[i];
    }
}

__global__ void gpuElementwiseMultScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] * n;
    }
}

__global__ void gpuRelu(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] > 0 ? data1[i] : 0;
    }
}

__global__ void gpuElementwiseDivision(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] / data2[i];
    }
}

__global__ void gpuElementwiseDivisionScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] / n;
    }
}

__global__ void gpuElementwiseDivisionScalar2(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n / data1[i];
    }
}

__global__ void gpuBinarize(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] > 0 ? 1 : 0;
    }
}

__global__ void gpuMatmul2d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t data1Dims1, size_t data2Dims1){
    for(size_t i = blockIdx.x  * blockDim.x + threadIdx.x; i < retDims0; i += NUMBLOCKS2D * NUMTHREADS2D){
        for(size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < retDims1; j += NUMBLOCKS2D * NUMTHREADS2D){
            ret[i * retDims0 + j] = 0;
            for(size_t k = 0; k < data1Dims1; ++k){
                ret[i * retDims1 + j] += data1[i * data1Dims1 + k] * data2[k * data2Dims1 + j];
            }
        }
    }
}

__global__ void gpuMatmul3d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t retDims2, size_t data1Dims1, size_t data1Dims2, size_t data2Dims1){
    for(size_t b = blockIdx.x * blockDim.x + threadIdx.x; b < retDims0; b += NUMBLOCKS3D * NUMTHREADS3D){
        for(size_t i = blockIdx.y * blockDim.y + threadIdx.y; i < retDims1; i += NUMBLOCKS3D * NUMTHREADS3D){
            for(size_t j = blockIdx.z * blockDim.z + threadIdx.z; j < retDims2; j += NUMBLOCKS3D * NUMTHREADS3D){
                ret[b * retDims2 * retDims1 + i * retDims1 + j] = 0;
                for(size_t k = 0; k < data1Dims2; ++k){
                    ret[b * retDims2 * retDims1 + i * retDims2 + j] += data1[b * data1Dims2 * data1Dims1 + i * data1Dims2 + k] * data2[k * data2Dims1 + j];
                }
            }
        }
    }
}

__global__ void gpuTranspose2d(double * ret, double * data1, size_t retDims0, size_t retDims1){
    for(size_t i = blockIdx.x  * blockDim.x + threadIdx.x; i < retDims0; i += NUMBLOCKS2D * NUMTHREADS2D){
        for(size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < retDims1; j += NUMBLOCKS2D * NUMTHREADS2D){
            ret[j * retDims0 + i] = data1[i * retDims1 + j];
        }
    }
}

__global__ void gpuTranspose3d(double * ret, double * data1, size_t retDims0, size_t retDims1, size_t retDims2){
    for(size_t b = blockIdx.x * blockDim.x + threadIdx.x; b < retDims0; b += NUMBLOCKS3D * NUMTHREADS3D){
        for(size_t i = blockIdx.y * blockDim.y + threadIdx.y; i < retDims1; i += NUMBLOCKS3D * NUMTHREADS3D){
            for(size_t j = blockIdx.z * blockDim.z + threadIdx.z; j < retDims2; j += NUMBLOCKS3D * NUMTHREADS3D){
                ret[b * retDims1 * retDims2 + j * retDims1 + i] = data1[b * retDims1 * retDims2 + i * retDims2 + j];
            }
        }
    }
}

__global__ void gpuReduceSum(double * ret, double * data1, size_t dataLen){
    // kernel must be started with <<<1,1>>>
    ret[0] = 0;
    for(size_t i = 0; i < dataLen; ++i){
        ret[0] += data1[i];
    }
}

void gpuSNeg(double * ret, double * data1, size_t dataLen)
{gpuNeg<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, dataLen);}

void gpuSAdd(double * ret, double * data1, double * data2, size_t dataLen)
{gpuAdd<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, data2, dataLen);}

void gpuSAddScalar(double * ret, double * data1, double n, size_t dataLen)
{gpuAddScalar<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSSubtract(double * ret, double * data1, double * data2, size_t dataLen)
{gpuSubtract<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, data2, dataLen);}

void gpuSSubtractScalar(double * ret, double * data1, double n, size_t dataLen)
{gpuSubtractScalar<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSScalarSubtract(double * ret, double * data1, double n, size_t dataLen)
{gpuScalarSubtract<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSPow(double * ret, double * data1, double n, size_t dataLen)
{gpuPow<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSZeroes(double * ret, size_t dataLen)
{gpuZeroes<<<NUMBLOCKS, NUMTHREADS>>>(ret, dataLen);}

void gpuSOnes(double * ret, size_t dataLen)
{gpuOnes<<<NUMBLOCKS, NUMTHREADS>>>(ret, dataLen);}

void gpuSFill(double * ret, double n, size_t dataLen)
{gpuFill<<<NUMBLOCKS, NUMTHREADS>>>(ret, n, dataLen);}

void gpuSElementwiseMult(double * ret, double * data1, double * data2, size_t dataLen)
{gpuElementwiseMult<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, data2, dataLen);}

void gpuSElementwiseMultScalar(double * ret, double * data1, double n, size_t dataLen)
{gpuElementwiseMultScalar<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSElementwiseDivision(double * ret, double * data1, double * data2, size_t dataLen)
{gpuElementwiseDivision<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, data2, dataLen);}

void gpuSElementwiseDivisionScalar(double * ret, double * data1, double n, size_t dataLen)
{gpuElementwiseDivisionScalar<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSElementwiseDivisionScalar2(double * ret, double * data1, double n, size_t dataLen)
{gpuElementwiseDivisionScalar2<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, n, dataLen);}

void gpuSRelu(double * ret, double * data1, size_t dataLen)
{gpuRelu<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, dataLen);}

void gpuSBinarize(double * ret, double * data1, size_t dataLen)
{gpuBinarize<<<NUMBLOCKS, NUMTHREADS>>>(ret, data1, dataLen);}

void gpuSMatmul2d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t data1Dims1, size_t data2Dims1)
{gpuMatmul2d<<<numblocks2dDIM, numthreads2dDIM>>>(ret, data1, data2, retDims0, retDims1, data1Dims1, data2Dims1);}

void gpuSMatmul3d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t retDims2, size_t data1Dims1, size_t data1Dims2, size_t data2Dims1)
{gpuMatmul3d<<<numblocks3dDIM, numthreads3dDIM>>>(ret, data1, data2, retDims0, retDims1, retDims2, data1Dims1, data1Dims2, data2Dims1);}

void gpuSTranspose2d(double * ret, double * data1, size_t retDims0, size_t retDims1)
{gpuTranspose2d<<<numblocks2dDIM, numthreads2dDIM>>>(ret, data1, retDims0, retDims1);}

void gpuSTranspose3d(double * ret, double * data1, size_t retDims0, size_t retDims1, size_t retDims2)
{gpuTranspose3d<<<numblocks3dDIM, numthreads3dDIM>>>(ret, data1, retDims0, retDims1, retDims2);}

void gpuSReduceSum(double * ret, double * data1, size_t dataLen)
{gpuReduceSum<<<1, 1>>>(ret, data1, dataLen);}
