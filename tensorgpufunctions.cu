#include "hip/hip_runtime.h"
#include "tensorgpufunctions.h"

__global__ void gpuNeg(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = -data1[i];
    }
}

__global__ void gpuAdd(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] + data2[i];
    }
}

__global__ void gpuAddScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] + n;
    }
}

__global__ void gpuSubtract(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] - data2[i];
    }
}

__global__ void gpuSubtractScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] - n;
    }
}

__global__ void gpuScalarSubtract(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n - data1[i];
    }
}

__global__ void gpuPow(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = pow(data1[i], n);
    }
}

__global__ void gpuZeroes(double * ret, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = 0;
    }
}

__global__ void gpuOnes(double * ret, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = 1;
    }
}

__global__ void gpuFill(double * ret, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n;
    }
}

__global__ void gpuElementwiseMult(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] * data2[i];
    }
}

__global__ void gpuElementwiseMultScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] * n;
    }
}

__global__ void gpuRelu(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] > 0 ? data1[i] : 0;
    }
}

__global__ void gpuElementwiseDivision(double * ret, double * data1, double * data2, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] / data2[i];
    }
}

__global__ void gpuElementwiseDivisionScalar(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] / n;
    }
}

__global__ void gpuElementwiseDivisionScalar2(double * ret, double * data1, double n, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = n / data1[i];
    }
}

__global__ void gpuBinarize(double * ret, double * data1, size_t dataLen){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < dataLen; i += NUMBLOCKS * NUMTHREADS){
        ret[i] = data1[i] > 0 ? 1 : 0;
    }
}

__global__ void gpuMatmul2d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t data1Dims1, size_t data2Dims1){
    for(size_t i = blockIdx.x  * blockDim.x + threadIdx.x; i < retDims0; i += NUMBLOCKS2D * NUMTHREADS2D){
        for(size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < retDims1; j += NUMBLOCKS2D * NUMTHREADS2D){
            ret[i * retDims0 + j] = 0;
            for(size_t k = 0; k < data1Dims1; ++k){
                ret[i * retDims1 + j] += data1[i * data1Dims1 + k] * data2[k * data2Dims1 + j];
            }
        }
    }
}

__global__ void gpuMatmul3d(double * ret, double * data1, double * data2, size_t retDims0, size_t retDims1, size_t retDims2, size_t data1Dims1, size_t data1Dims2, size_t data2Dims1){
    for(size_t b = blockIdx.x * blockDim.x + threadIdx.x; b < retDims0; b += NUMBLOCKS3D * NUMTHREADS3D){
        for(size_t i = blockIdx.y * blockDim.y + threadIdx.y; i < retDims1; i += NUMBLOCKS3D * NUMTHREADS3D){
            for(size_t j = blockIdx.z * blockDim.z + threadIdx.z; j < retDims2; j += NUMBLOCKS3D * NUMTHREADS3D){
                ret[b * retDims2 * retDims1 + i * retDims1 + j] = 0;
                for(size_t k = 0; k < data1Dims2; ++k){
                    ret[b * retDims2 * retDims1 + i * retDims2 + j] += data1[b * data1Dims2 * data1Dims1 + i * data1Dims2 + k] * data2[k * data2Dims1 + j];
                }
            }
        }
    }
}

__global__ void gpuTranspose2d(double * ret, double * data1, size_t retDims0, size_t retDims1){
    for(size_t i = blockIdx.x  * blockDim.x + threadIdx.x; i < retDims0; i += NUMBLOCKS2D * NUMTHREADS2D){
        for(size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < retDims1; j += NUMBLOCKS2D * NUMTHREADS2D){
            ret[j * retDims0 + i] = data1[i * retDims1 + j];
        }
    }
}

__global__ void gpuTranspose3d(double * ret, double * data1, size_t retDims0, size_t retDims1, size_t retDims2){
    for(size_t b = blockIdx.x * blockDim.x + threadIdx.x; b < retDims0; b += NUMBLOCKS3D * NUMTHREADS3D){
        for(size_t i = blockIdx.y * blockDim.y + threadIdx.y; i < retDims1; i += NUMBLOCKS3D * NUMTHREADS3D){
            for(size_t j = blockIdx.z * blockDim.z + threadIdx.z; j < retDims2; j += NUMBLOCKS3D * NUMTHREADS3D){
                ret[b * retDims1 * retDims2 + j * retDims1 + i] = data1[b * retDims1 * retDims2 + i * retDims2 + j];
            }
        }
    }
}

void gpuReduceSum(double * ret, double * data1, size_t dataLen){
    // kernel must be started with <<<1,1>>>
    ret[0] = 0;
    for(size_t i = 0; i < dataLen; ++i){
        ret[0] += data1[i];
    }
}
